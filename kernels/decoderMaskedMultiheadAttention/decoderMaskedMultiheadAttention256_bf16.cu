#include "hip/hip_runtime.h"


#include "decoderMaskedMultiheadAttentionLaunch.h"

namespace bitfusion
{
namespace kernels
{

namespace
{
auto constexpr kSizePerHead = 256;
} // namespace

namespace mmha
{

#ifdef ENABLE_BF16
INSTANTIATE_MMHA_LAUNCHERS(__hip_bfloat16, kSizePerHead)
#endif

} // namespace mmha

} // namespace kernels
} // namespace bitfusion
