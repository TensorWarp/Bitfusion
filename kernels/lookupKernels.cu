#include "hip/hip_runtime.h"


#include "../common/cudaTypeUtils.cuh"
#include "lookupKernels.h"

using namespace bitfusion::common;

namespace bitfusion
{
namespace kernels
{

template <typename T, typename Idx>
__global__ void lookup_kernel(T* output, const Idx* input, const T* weight, const Idx batch_size, const Idx offset,
    const Idx size, const int n_embed)
{
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < batch_size * n_embed;
         index += blockDim.x * gridDim.x)
    {
        const int word_index = input[index / n_embed] - offset;
        const int col_index = index % n_embed;
        T embedding;
        if (word_index < 0 || word_index >= size)
        {
            embedding = T(0.f);
        }
        else
        {
            embedding = weight[word_index * n_embed + col_index];
        }
        output[index] = embedding;
    } // end for index
}

template <typename T, typename Idx>
void invokeLookUp(T* out, const Idx* input, const T* weight, const Idx batch_size, const Idx offset, const Idx size,
    const int n_embed, hipStream_t stream)
{
    dim3 grid(min(batch_size, 65536));
    dim3 block(min(n_embed, 512));
    lookup_kernel<T, Idx><<<grid, block, 0, stream>>>(out, input, weight, batch_size, offset, size, n_embed);
}

#define INSTANTIATE_LOOK_UP(T, Idx)                                                                                    \
    template void invokeLookUp<T, Idx>(T * out, const Idx* input, const T* weight, const Idx batch_size,               \
        const Idx offset, const Idx size, const int n_embed, hipStream_t stream)

INSTANTIATE_LOOK_UP(float, int);
INSTANTIATE_LOOK_UP(half, int);

#ifdef ENABLE_BF16
INSTANTIATE_LOOK_UP(__hip_bfloat16, int);
#endif

} // namespace kernels
} // namespace bitfusion
