#include "hip/hip_runtime.h"


#include "../../../kernels/cutlass_kernels/int8_gemm/int8_gemm_template.h"

namespace bitfusion
{
namespace kernels
{
namespace cutlass_kernels
{

#ifdef ENABLE_BF16
template class CutlassInt8GemmRunner<__hip_bfloat16>;
#endif

} // namespace cutlass_kernels
} // namespace kernels
} // namespace bitfusion
