#include "hip/hip_runtime.h"


#include "../../cutlass_kernels/moe_gemm/moe_gemm_kernels_template.h"

namespace bitfusion
{
#ifdef ENABLE_BF16
template class MoeGemmRunner<__hip_bfloat16, uint8_t>;
#endif
} // namespace bitfusion
