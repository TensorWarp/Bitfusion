#include "hip/hip_runtime.h"


#include "../../cutlass_kernels/moe_gemm/moe_gemm_kernels_template.h"

namespace bitfusion
{
#ifdef ENABLE_BF16
template class MoeGemmRunner<__hip_bfloat16, cutlass::uint4b_t>;
#endif
} // namespace bitfusion
