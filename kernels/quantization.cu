#include "hip/hip_runtime.h"


#include "../common/assert.h"
#include "../common/cudaTypeUtils.cuh"
#include "../common/cudaUtils.h"
#include "../common/reduceKernelUtils.cuh"
#include "quantization.h"

using namespace bitfusion::common;

namespace bitfusion
{
namespace kernels
{

__global__ void quantizedKernel(char4* dst, const float4* src, const int64_t sizeDiv4, const float* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        const float scale = __ldg(scalePtr);
        char4 tmp;
        const float4 floatTmp = __ldg(src + idx);
        tmp.x = cuda_cast<int8_t>(floatTmp.x * scale);
        tmp.y = cuda_cast<int8_t>(floatTmp.y * scale);
        tmp.z = cuda_cast<int8_t>(floatTmp.z * scale);
        tmp.w = cuda_cast<int8_t>(floatTmp.w * scale);
        dst[idx] = tmp;
    }
}

__global__ void quantizedKernel(char4* dst, const half2* src, const int64_t sizeDiv4, const float* scalePtr)
{
    for (int64_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < sizeDiv4; idx += blockDim.x * gridDim.x)
    {
        const float scale = __ldg(scalePtr);
        char4 tmp;
        int srcId = idx << 1;

        const uint2 h2 = __ldg(reinterpret_cast<const uint2*>(src + srcId));

        const half2 half2Tmp = reinterpret_cast<const half2&>(h2.x);
        const half2 half2Tmp2 = reinterpret_cast<const half2&>(h2.y);

        tmp.x = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.x) * scale);
        tmp.y = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp.y) * scale);
        tmp.z = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.x) * scale);
        tmp.w = cuda_cast<int8_t>(cuda_cast<float>(half2Tmp2.y) * scale);
        dst[idx] = tmp;
    }
}

template <typename T>
void invokeQuantization(
    int8_t* dst, const T* src, const int64_t size, const float* scalePtr, hipStream_t stream, int maxGridSize)
{
    CHECK_WITH_INFO(size % 4 == 0, "[ERROR][invokeQuantization] size should be a multiple of 4.\n");

    int numBlocks{static_cast<int>((size + 255) / 256)};
    dim3 grid(std::min(numBlocks, maxGridSize));
    CHECK_WITH_INFO(grid.x <= maxGridSize, "[ERROR][invokeQuantization] grid max size is exceeded\n");
    dim3 block(64);
    if (std::is_same_v<T, float>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (const float4*) src, size / 4, scalePtr);
    }
    else if (std::is_same_v<T, half>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (const half2*) src, size / 4, scalePtr);
    }
}

template void invokeQuantization<float>(
    int8_t* dst, const float* src, const int64_t size, const float* scalePtr, hipStream_t stream, int maxGridSize);

template void invokeQuantization<half>(
    int8_t* dst, const half* src, const int64_t size, const float* scalePtr, hipStream_t stream, int maxGridSize);

template <typename T>
__global__ void perTokenQuantization(
    int8_t* dst, const T* src, const int64_t numRows, const int64_t numCols, float* scalePtr)
{
    const T* srcRow = src + blockIdx.x * numCols;
    int8_t* dstRow = dst + blockIdx.x * numCols;

    T localMax = 1e-6f;
    for (int i = threadIdx.x; i < numCols; i += blockDim.x)
    {
        localMax = cuda_max(localMax, cuda_abs(srcRow[i]));
    }
    const float rowMax = blockAllReduceMax(cuda_cast<float>(localMax));

    if (threadIdx.x == 0)
    {
        scalePtr[blockIdx.x] = rowMax / 127.f;
    }

    const float scaleOrigQuant = 127.f / rowMax;
    for (int i = threadIdx.x; i < numCols; i += blockDim.x)
    {
        dstRow[i] = cuda_cast<int8_t>(cuda_cast<float>(srcRow[i]) * scaleOrigQuant);
    }
}

template <typename T>
void invokePerTokenQuantization(
    int8_t* dst, const T* src, const int64_t numRows, const int64_t numCols, float* scalePtr, hipStream_t stream)
{
    // each block is responsible for a single row
    const dim3 block(512);
    const dim3 grid(numRows);

    perTokenQuantization<<<grid, block, 0, stream>>>(dst, src, numRows, numCols, scalePtr);
}

#define INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(T)                                                                   \
    template void invokePerTokenQuantization(                                                                          \
        int8_t* dst, const T* src, const int64_t numRows, const int64_t numCols, float* scalePtr, hipStream_t stream)

INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16);
#endif

} // namespace kernels
} // namespace bitfusion
